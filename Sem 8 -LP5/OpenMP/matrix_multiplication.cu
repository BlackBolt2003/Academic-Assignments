#include<iostream>
#include<hip/hip_runtime.h>
using namespace std;

__global__ void matrix_mul(int *a, int *b, int *c, int size){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < size && col < size){
        int sum = 0;
        for(int i=0; i<size; i++){
            sum += a[row*size + i] * b[i*size + col];
        }
        c[row*size + col] = sum;
    }
}

void init_matrix(int *matrix, int size){
    for(int i=0; i<size*size; i++){
        matrix[i] = rand() % 10;
    }
}

void print_matrix(int *matrix, int size){
    for(int row=0; row<size; row++){
        for(int col=0; col<size; col++){
            cout << matrix[row*size + col] << " ";
        }
        cout << endl;
    }
    cout << endl;
}

int main(){
    int n = 4;
    int *a, *b, *c;
    int matrixSize = n * n;

    size_t matrixBytes = matrixSize * sizeof(int);

    a = new int[matrixSize];
    b = new int[matrixSize];
    c = new int[matrixSize];

    init_matrix(a, n);
    cout << "Matrix A:\n";
    print_matrix(a, n);

    init_matrix(b, n);
    cout << "Matrix B:\n";
    print_matrix(b, n);

    int *da, *db, *dc;
    hipMalloc(&da, matrixBytes);
    hipMalloc(&db, matrixBytes);
    hipMalloc(&dc, matrixBytes);

    hipMemcpy(da, a, matrixBytes, hipMemcpyHostToDevice);
    hipMemcpy(db, b, matrixBytes, hipMemcpyHostToDevice);

    int THREADS = 16;
    int BLOCKS = (n + THREADS - 1) / THREADS;

    dim3 threads(THREADS, THREADS);
    dim3 blocks(BLOCKS, BLOCKS);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    matrix_mul<<<blocks, threads>>>(da, db, dc, n);
    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
        cout << "CUDA Error: " << hipGetErrorString(error) << endl;
    }
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    hipDeviceSynchronize();

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(c, dc, matrixBytes, hipMemcpyDeviceToHost);

    cout << "Resultant Matrix C:\n";
    print_matrix(c, n);

    cout << "\nKernel execution time: " << milliseconds << " ms" << endl;


    delete[] a;
    delete[] b;
    delete[] c;

    hipFree(da);
    hipFree(db);
    hipFree(dc);

    return 0;
}