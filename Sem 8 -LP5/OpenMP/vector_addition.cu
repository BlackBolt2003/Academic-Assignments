#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vector_addition(float *A, float *B, float *C, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        C[i] = A[i] + B[i];
    }
}

void initialize(float *arr, int n)
{
    for (int i = 0; i < n; i++)
    {
        arr[i] = rand() % n;
    }
}

int main()
{
    int n = 1000;
    size_t size = n * sizeof(float);

    float *A = new float[n];
    float *B = new float[n];
    float *C = new float[n];

    initialize(A, n);
    initialize(B, n);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    vector_addition<<<1, n>>>(d_A, d_B, d_C, n);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    printf("Result Vector C:\t");
    for (int i = 0; i < n; i++)
    {
        printf("%f ", C[i]);
    }
    printf("\n");

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}